/**
 * \file pipeline.cu
 * \brief type declarations vision pipelines.
 * \copyright 2015, Juan David Adarve, ANU. See AUTHORS for more details
 * \license 3-clause BSD, see LICENSE for more details
 */

#include <exception>
#include <iostream>

#include "flowfilter/gpu/error.h"
#include "flowfilter/gpu/pipeline.h"


namespace flowfilter {
    namespace gpu {

        //#################################################
        // Stage
        //#################################################
        Stage::Stage() :
            Stage(0) {
        }

        Stage::Stage(hipStream_t stream) {
            checkError(hipSetDevice(0));
            __stream = stream;
            __elapsedTime = 0.0f;
            __referenceCounter = std::make_shared<int>(0);

            checkError(hipEventCreate(&__start));
            checkError(hipEventCreate(&__stop));

            // if(startErr != hipSuccess || stopErr != hipSuccess) {
            //     std::cerr << "Stage::Stage(): error creating CUDA events: "
            //     << hipGetErrorString(startErr) << " - "
            //     << hipGetErrorString(stopErr) << std::endl;

            //     throw std::exception();
            // }
        }

        Stage::~Stage() {

            // std::cout << "Stage::~Stage(): " << __referenceCounter.use_count() << std::endl;

            if(__referenceCounter.use_count() == 1) {

                checkError(hipEventDestroy(__start));
                checkError(hipEventDestroy(__stop));

                // if(startErr != hipSuccess || stopErr != hipSuccess) {
                //     std::cerr << "Stage::Stage(): error destroying CUDA events: "
                //     << hipGetErrorString(startErr) << " - "
                //     << hipGetErrorString(stopErr) << std::endl;

                //     throw std::exception();
                // }    
            }
        }


        void Stage::startTiming() {
            checkError(hipEventRecord(__start, __stream));
            // hipError_t startErr = hipGetLastError();
            // if(startErr != hipSuccess) {
            //     std::cerr << "ERROR: Stage::startTiming(): error starting timing: "
            //     << hipGetErrorString(startErr) << std::endl;
            // }
        }

        void Stage::stopTiming() {
            checkError(hipEventRecord(__stop, __stream));
            checkError(hipEventSynchronize(__stop));
            checkError(hipEventElapsedTime(&__elapsedTime, __start, __stop));

            hipError_t stopErr = hipGetLastError();
            if(stopErr != hipSuccess) {
                std::cerr << "ERROR: Stage::startTiming(): error stoping timing: "
                << hipGetErrorString(stopErr) << std::endl;
            }
        }

        /**
         * \brief return computation elapsed time in milliseconds
         */
        float Stage::elapsedTime() const {
            return __elapsedTime;
        }


        //#################################################
        // EmptyStage
        //#################################################
        EmptyStage::EmptyStage() :
            Stage() {

            // nothing to do
        }

        EmptyStage::~EmptyStage() {
            // nothing to do
        }

        void EmptyStage::configure() {
            // nothing to do...
        }

        void EmptyStage::compute() {
            
            startTiming();

            // no operation to be performed

            stopTiming();
        }

    };
};