/**
 * \file image.cu
 * \brief type declarations for GPU image buffers.
 * \copyright 2015, Juan David Adarve, ANU. See AUTHORS for more details
 * \license 3-clause BSD, see LICENSE for more details
 */

#include <cstring>
#include <string>
#include <iostream>
#include <exception>
#include <stdexcept>

#include "flowfilter/gpu/image.h"
#include "flowfilter/gpu/gpu_deleter.h"
#include "flowfilter/gpu/error.h"

namespace flowfilter {
namespace gpu {

//#################################################
// GPUImage
//#################################################
GPUImage::GPUImage() {
    __width = 0;
    __height = 0;
    __depth = 0;
    __pitch = 0;
    __itemSize = 0;
}

GPUImage::GPUImage(const int height, const int width,
    const int depth, const int itemSize) {

    __height = height;
    __width = width;
    __depth = depth;
    __itemSize = itemSize;

    // allocate memory in GPU space
    allocate();
}

GPUImage::~GPUImage() {

    // nothing to do
    // device buffer is released by gpu_deleter
    // std::cout << "GPUImage::~GPUImage(): [" << 
    //     __height << ", " << __width << ", " << __depth << "] : " << __ptr_dev.use_count() << std::endl;
}

int GPUImage::height() const {
    return __height;
}

int GPUImage::width() const {
    return __width;
}

int GPUImage::depth() const {
    return __depth;
}

int GPUImage::pitch() const {
    return __pitch;
}

int GPUImage::itemSize() const {
    return __itemSize;
}

void* GPUImage::data() {
    return __ptr_dev.get();
}

std::shared_ptr<void> GPUImage::getDeviceBuffer() {
    return __ptr_dev;
}


void GPUImage::upload(flowfilter::image_t& img) {

    // check if device memory is allocated
    if(!__ptr_dev) {

        // set resolution to input image
        __width = img.width;
        __height = img.height;
        __depth = img.depth;
        __itemSize = img.itemSize;

        // allocate memory
        allocate();
    }

    // compare shapes
    if(compareShape(img)) {

        // print first 5 elements of img
        // for(int i = 0; i < 5; i ++) {
        //     std::cout << i << ": " << (int)static_cast<unsigned char*>(img.data)[i] << std::endl;
        // }

        // issue synchronous memory copy
        checkError(hipMemcpy2D(__ptr_dev.get(), __pitch, img.data, img.pitch,
            __width*__depth*__itemSize, __height,
            hipMemcpyHostToDevice));

        // TODO: add support for asynchronous copy

    } else {

        std::cerr << "ERROR: GPUImage::upload(): shapes do not match."
            << "required: [" << __height << ", " << __width << ", " << __depth << "][" << __itemSize << "], passed: "
            << "[" << img.height << ", " << img.width << ", " << img.depth << "][" << img.itemSize << "]" << std::endl;

        throw std::invalid_argument("GPUImage::upload(): shapes do not match. Required: [" +
            std::to_string(__height) + ", " + std::to_string(__width) + ", " + std::to_string(__depth) + "][" + std::to_string(__itemSize) + "], passed: [" +
            std::to_string(img.height) + ", " + std::to_string(img.width) + ", " + std::to_string(img.depth) + "][" + std::to_string(img.itemSize) + "]");
    }
}

void GPUImage::download(flowfilter::image_t& img) const {

    if(!__ptr_dev) {
        std::cerr << "ERROR: GPUImage::download(): unallocated image" << std::endl;
        return; // TODO: throw exception
    }

    if(compareShape(img)) {

        // issue synchronous memory copy
        checkError(hipMemcpy2D(img.data, img.pitch, __ptr_dev.get(), __pitch,
            __width*__depth*__itemSize, __height, hipMemcpyDeviceToHost));

        // print first 5 elements of img
        // for(int i = 0; i < 5; i ++) {
        //     std::cout << i << ": " << static_cast<float*>(img.data)[i] << std::endl;
        // }

    } else {
        std::cerr << "ERROR: GPUImage::download(): shapes do not match."
            << "required: [" << __height << ", " << __width << ", " << __depth << "][" << __itemSize << "], passed: "
            << "[" << img.height << ", " << img.width << ", " << img.depth << "][" << img.itemSize << "]" << std::endl;

        throw std::invalid_argument("GPUImage::download(): shapes do not match. Required: [" +
            std::to_string(__height) + ", " + std::to_string(__width) + ", " + std::to_string(__depth) + "][" + std::to_string(__itemSize) + "], passed: [" +
            std::to_string(img.height) + ", " + std::to_string(img.width) + ", " + std::to_string(img.depth) + "][" + std::to_string(img.itemSize) + "]");
    }
}

void GPUImage::copyFrom(GPUImage& img) {

    if(compareShapeGPU(img)) {

        // issue synchronous memory copy
        checkError(hipMemcpy2D(__ptr_dev.get(), __pitch, 
            img.__ptr_dev.get(), img.__pitch,
            __width*__depth*__itemSize, __height, hipMemcpyDeviceToDevice));

    } else {
        std::cerr << "ERROR: GPUImage::copyFrom(): shapes do not match."
            << "required: [" << __height << ", " << __width << ", " << __depth << "][" << __itemSize << "], passed: "
            << "[" << img.__height << ", " << img.__width << ", " << img.__depth << "][" << img.__itemSize << "]" << std::endl;

        throw std::invalid_argument("GPUImage::copyFrom(): shapes do not match. Required: [" +
            std::to_string(__height) + ", " + std::to_string(__width) + ", " + std::to_string(__depth) + "][" + std::to_string(__itemSize) + "], passed: [" +
            std::to_string(img.__height) + ", " + std::to_string(img.__width) + ", " + std::to_string(img.__depth) + "][" + std::to_string(img.__itemSize) + "]");
    }
}

void GPUImage::clear() {

    checkError(hipMemset2D(__ptr_dev.get(), __pitch,
        0, __width*__depth*__itemSize, __height));
}


void GPUImage::allocate() {

    // std::cout << "GPUImage::allocate()" << std::endl;

    void* buffer_dev = nullptr;
    checkError(hipMallocPitch(&buffer_dev, &__pitch,
        __width*__depth*__itemSize, __height));

    // create a new shared pointer
    __ptr_dev = std::shared_ptr<void> (buffer_dev, gpu_deleter<void>());

    // std::cout << "\tpitch: " << __pitch << std::endl;

    // if(err != hipSuccess) {
    //     std::cerr << "ERROR: GPUImage device memory allocation: " << hipGetErrorString(err) << std::endl;
    //     // TODO: throw exception?
    //     throw std::bad_alloc("GPUImage::allocate(): device memory allocation error: " + hipGetErrorString(err));
    // }
}

bool GPUImage::compareShape(const flowfilter::image_t& img) const {

    return __height == img.height &&
        __width == img.width &&
        __depth == img.depth &&
        __itemSize == img.itemSize;
}

bool GPUImage::compareShapeGPU(const flowfilter::gpu::GPUImage& img) const {

    return __height == img.height() &&
        __width == img.width() &&
        __depth == img.depth() &&
        __pitch == img.pitch() &&
        __itemSize == img.itemSize();
}



//#################################################
// GPUTexture
//#################################################
GPUTexture::GPUTexture() {

    // texture object is not valid
    __validTexture = false;
    __refCounter = std::make_shared<int>(0);
}

GPUTexture::GPUTexture( GPUImage& img, hipChannelFormatKind format) :
    GPUTexture(img, format, hipAddressModeClamp, 
        hipFilterModePoint, hipReadModeElementType, false) {
}

GPUTexture::GPUTexture( GPUImage& img,
                        hipChannelFormatKind format,
                        hipTextureReadMode readMode) : 
    GPUTexture(img, format, hipAddressModeClamp,
        hipFilterModePoint, readMode, false) {

}

GPUTexture::GPUTexture( flowfilter::gpu::GPUImage& img,
                        hipChannelFormatKind format,
                        hipTextureReadMode readMode,
                        const bool normalizedCoords) : 
    GPUTexture(img, format, hipAddressModeClamp, hipFilterModePoint,
        readMode, normalizedCoords) {

}

GPUTexture::GPUTexture( GPUImage& img,
                        hipChannelFormatKind format,
                        hipTextureAddressMode addressMode,
                        hipTextureFilterMode filterMode,
                        hipTextureReadMode readMode,
                        const bool normalizedCoords) {

    __refCounter = std::make_shared<int>(0);

    // hold input image
    __image = img;

    // configure CUDA texture
    configure(format, addressMode, filterMode, readMode, normalizedCoords);
}

GPUTexture::~GPUTexture() {

    // std::cout << "GPUTexture::~GPUTexture(): " <<  __refCounter.use_count() << " : " << __texture << std::endl;


    // only attempts to destroy the texture if the creation
    // was successful
    if(__refCounter.use_count() == 1) {
        if(__validTexture) {
            checkError(hipDestroyTextureObject(__texture));    
        }
    }

    // __image destructor is called automatically and
    // devide buffer is released only if it's not being
    // shared in any other part of the program.
}

hipTextureObject_t GPUTexture::getTextureObject() {
    return __texture;
}

GPUImage GPUTexture::getImage() {
    return __image;
}

void GPUTexture::configure( hipChannelFormatKind format,
                            hipTextureAddressMode addressMode,
                            hipTextureFilterMode filterMode,
                            hipTextureReadMode readMode,
                            const bool normalizedCoords) {
    
    __validTexture = false;

    int channels = __image.depth();
    if(channels > 4) {
        std::cerr << "ERROR: GPUTexture::configure(): image channels greater than 4: " << channels << std::endl;
        throw std::invalid_argument("GPUTexture::configure(): image channels greater than 4, got: " + std::to_string(channels));
    }

    // bit width of element
    int bitWidth = 8 * __image.itemSize();

    // bit width of each channel
    int w1 = bitWidth;  // there is at least one channel
    int w2 = channels >= 2? bitWidth : 0;
    int w3 = channels >= 3? bitWidth : 0;
    int w4 = channels == 4? bitWidth : 0;

    // std::cout << "GPUTexture::configure(): channel width: [" << w1 << ", " << w2 << ", " << w3 << ", " << w4 << "]" << std::endl;
    // std::cout << "GPUTexture::configure(): [" << __image.height() << ", " << __image.width() << ", " << __image.depth() << "] size: " << __image.itemSize() << " pitch: " << __image.pitch() << std::endl;

    // channel descriptor
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(w1, w2, w3, w4, format);

    // texture descriptor
    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = addressMode;
    texDesc.addressMode[1] = addressMode;
    texDesc.filterMode = filterMode;
    texDesc.readMode = readMode;
    texDesc.normalizedCoords = normalizedCoords;

    // texture buffer descriptor
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypePitch2D;
    resDesc.res.pitch2D.desc = channelDesc;
    resDesc.res.pitch2D.devPtr = __image.data();
    resDesc.res.pitch2D.pitchInBytes = __image.pitch();
    resDesc.res.pitch2D.width = __image.width();
    resDesc.res.pitch2D.height = __image.height();

    // creates texture
    checkError(hipCreateTextureObject(&__texture, &resDesc, &texDesc, NULL));
    __validTexture = true;

    // hipError_t err = hipCreateTextureObject(&__texture, &resDesc, &texDesc, NULL);

    // std::cout << "GPUTexture::configure(): texture ID: " << __texture << std::endl;
    // if(err == hipSuccess) {
    //     __validTexture = true;
    // } else {

    //     std::cerr << "ERROR: GPUTexture::configure(): texture creation: "
    //         << hipGetErrorString(err) << std::endl;

    //     __validTexture = false;
    // }
}

}; // namespace gpu
}; // namespace flowfilter
